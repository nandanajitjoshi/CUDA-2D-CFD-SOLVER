
/*@Author : Nandan Joshi

* Implementation of a bicongugate Gradient asolver for linear equations
* Uses matrices in a sparse format
* Input matrix is in Compressed Space Row (CSR) format
* cuSparse allows matrix multiplications only in BSR (Block Spase Row) format
* CSR Converted to a BSR format within this code
* Details about CSR/BSR format : https://docs.nvidia.com/cuda/cusparse/index.html
*/


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include "helper_functions.h"  
#include "hip/hip_runtime_api.h"       



/* Function solves a linear system using BCG
* Adopted from https://en.wikipedia.org/wiki/Biconjugate_gradient_stabilized_method
* Input CSR matrix needs to be converted to BSR first
* @param X: Holds the solution of BCG - passed by ref
* @param  RHS : Holds the RHS of the linear system
* @param rowPtr, colPtr, val : row offsets, col indices, and values of CSR storage
* @param mb,nb : Rows, Cols of the modified matrix in BSR
* @param nnzb : Number of non zero blocks in BSR
* @param vecSize : Size of the solution vector in BSR representation
* @param maxIt : Maximum no of iterations
*/

void BCGSolve(double* X, double* RHS, int* rowBSR, int* colBSR, double*valBSR, 
        hipsparseHandle_t handle,  hipsparseMatDescr_t descr_coeff, 
        hipblasHandle_t handleBlas, 
        int mb, int nb, int nnzb, int vecSize, int maxit  ){

    /*Initialize variables*/


    double *R, *V, *T, *rw, *p;
    int dimBlock = 2;  //Block dimension hardcoded to 2

    double alpha = 1 ;
    double beta ;
    double alph;
    double bet;
    double omega = 1;
    double residual1 = 0;
    double residual2 = 0;
    double temp = 1;
    double rhop = 1; 
    double rho = 1;

    checkCudaErrors(hipMalloc((double **)&R, (vecSize)*sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&p, (vecSize)*sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&rw, (vecSize)*sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&V, (vecSize)*sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&T, (vecSize)*sizeof(double)));
    checkCudaErrors(hipMemset(V, 0, vecSize)); 


    const hipsparseOperation_t trans_coeff  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseDirection_t dir_coeff = HIPSPARSE_DIRECTION_ROW;

    /*Solve*/

   /*Step 1 : r = b - A.x*/ 
    alph = -1;
    bet = 0;

    /*-Calculate -Ax*/
    checkCudaErrors(hipsparseDbsrmv(handle, dir_coeff, trans_coeff, mb, nb, nnzb, &alph,
                                    descr_coeff,valBSR, rowBSR, colBSR, dimBlock,X, 
                                    &bet, R));


    alph = 1;  
    /*Calculate b-Ax*/
    checkCudaErrors( hipblasDaxpy( handleBlas, vecSize, 
                                    &alph,RHS, 1,R, 1));

    /*Assign r = b-Ax*/
    // checkCudaErrors (hipMemcpy(Y, R,  (vecSize)*sizeof(double), 
    //                  hipMemcpyDeviceToHost ));



    //Step 2: Set p=r and \tilde{r}=r
    checkCudaErrors(hipblasDcopy(handleBlas,(vecSize), R, 1, p, 1));         //p=r
    checkCudaErrors(hipblasDcopy(handleBlas, (vecSize), R, 1, rw,1));        //\tilde{r}=r
    checkCudaErrors(hipblasDnrm2(handleBlas,(vecSize), R, 1, &residual1));   //Residual 



    /*/ Step 3: repeat until convergence */
    for (int i=0; i<maxit; i++){
        rhop = rho; 
        //Step 5.1 : Dot product (rw,r)
        checkCudaErrors(hipblasDdot ( handleBlas, vecSize, rw, 1, R, 1, &rho));


        if (i > 0){
            //Step 5.2: \beta = (\rho_{i} / \rho_{i-1}) ( \alpha / \omega )
            beta= (rho/rhop)*(alpha/omega);

            //Step 5.3: p = r + \beta (p - \omega v)
            // -omega*v
            omega = -omega; 
            checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                            &omega,V, 1,p, 1));

            //Reset omega
            omega = -omega;


            // beta * (p - omega*v)
            checkCudaErrors(hipblasDscal(handleBlas, vecSize, 
                                &beta,p, 1)); 

            //  r + beta*(p-omega*v)
            checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                        &alph,R, 1,p, 1)); 


            // checkCudaErrors (hipMemcpy(Y, p,  (vecSize)*sizeof(double), hipMemcpyDeviceToHost ));

        }
            
            //Step 5.4 : v = A*p

            //A*p
            checkCudaErrors(hipsparseDbsrmv(handle, dir_coeff, trans_coeff, mb, nb, nnzb, &alph,
                descr_coeff,valBSR, rowBSR, colBSR, dimBlock,p, 
                &bet, V));

            //Assign v = A*p
            // checkCudaErrors (hipMemcpy(Y, V,  (vecSize)*sizeof(double), hipMemcpyDeviceToHost ));

            /*Step 5.5 alpha = rho/ dot (\tilde{r}, p)*/

            //Store  dot (\tilde{r}, p) in alpha
            checkCudaErrors(hipblasDdot ( handleBlas, vecSize, rw, 1, V, 1, &alpha));
            //Update alpha = rho/alpha
            alpha = rho/alpha; 


            /*Step 5.6/ s = r - \alpha * v */

            // Calculate -\alpha*v
            alpha = -alpha; 
            checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                        &alpha,V, 1,R, 1));  //S is stored in R to save memory


            //Reset alpha
            alpha = -alpha; 

            /*Step 5.5 X = X + p*alpha*/
            checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                &alpha,p, 1,X, 1));

            /*Step 5.7 : Check the residual of s*/
            checkCudaErrors(hipblasDnrm2(handleBlas,(vecSize), R, 1, &residual2));

            if (residual2/residual1 < 1E-3){
                /*Converged*/
                break;
            }

        
        /*Step 5.9 T = A*s*/
        checkCudaErrors(hipsparseDbsrmv(handle, dir_coeff, trans_coeff, mb, nb, nnzb, &alph,
            descr_coeff,valBSR, rowBSR, colBSR, dimBlock,R, 
            &bet, T));



        /*Step 5.10 omega = (T.T)/(T.R)*/
        checkCudaErrors(hipblasDdot ( handleBlas, vecSize, T, 1, T, 1, &temp));  
        checkCudaErrors(hipblasDdot ( handleBlas, vecSize, R, 1, T, 1, &omega));  

        omega = omega/temp; 

        //Step 5.11 *x = h + omega *s*/  

        checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                    &omega,R, 1,X, 1));


        /*Step 5.13 r = s - omega * t*/

        omega = -omega; 
        checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                    &omega,T, 1,R, 1));
        //Reset omega
        omega = -omega; 


        /*Step 5.12 Check residual of R*/
        checkCudaErrors(hipblasDnrm2(handleBlas,(vecSize), R, 1, &residual2));

         printf ("\n Residual %f \n", residual2/residual1);

        if (residual2/residual1 < 1E-3){
            /*Converged*/
            break;
        }
    }


}


/* Helper code to reserve buffer space for CSR to BSR conversion
* This function also calculates no of nonzero blocks in BSR
* @param rowPtr, colPtr, val : row offsets, col indices, and values of CSR storage
* @param rowBSR : row offsets for corresponding BSR format - passed by reference
* @param nnzb : non zero blocks in BSR - passed by reference
* @param  rows : Rows/Cols of the sparse matrix
* @param dimBlock : Dimension of block to be used in BSR
* @return pBuffer : Pointer to buffer space reserved for matrix operation
*/
void* getBSRDims(hipsparseHandle_t handle, int* rowPtr, int* colPtr, double* val, 
        int* rowBSR, int*nnzb, 
        int rows,  int dimBlock
        ){


    hipsparseMatDescr_t descr_coeff;
    hipsparseMatDescr_t descr_coeff_2;
    
    int bufferSize = 0; 
    static void *pBuffer; 

    const hipsparseOperation_t trans_coeff  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseDirection_t dir_coeff = HIPSPARSE_DIRECTION_ROW;

    checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff));
    checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff, HIPSPARSE_INDEX_BASE_ZERO));
    checkCudaErrors(hipsparseSetMatType(descr_coeff, HIPSPARSE_MATRIX_TYPE_GENERAL));

    checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff_2));
    checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff_2, HIPSPARSE_INDEX_BASE_ZERO));
    checkCudaErrors(hipsparseSetMatType(descr_coeff_2, HIPSPARSE_MATRIX_TYPE_GENERAL));

    /* Obtain buffer size for CSR to BSR conversion*/
    checkCudaErrors(hipsparseDcsr2gebsr_bufferSize(handle, dir_coeff, rows, rows,  
    descr_coeff, 
    val, rowPtr, colPtr, 
    dimBlock, dimBlock, 
    &bufferSize));

    /*Reserve buffer space*/
    checkCudaErrors(hipMalloc((void**)&pBuffer, bufferSize));

    /*Get the number of nonzero blocks*/
    checkCudaErrors(hipsparseXcsr2gebsrNnz(handle, dir_coeff, rows, rows, 
    descr_coeff, 
    rowPtr, colPtr,
    descr_coeff_2, 
    rowBSR, dimBlock, dimBlock, 
    nnzb, pBuffer));

    return pBuffer; 

}


/* Solves a linear system of euqations using BCG
* Accepts coefficient matrix in a sparse CSR storage
* Converts CSR into BSR before solving
* @param rowPtr, colPtr, val : row offsets, col indices, and values of coefficients
* @param Soln :Solution to the equations
* @param RHS : Right hand side of the equations
* @param nz : Nonzero elements in coeffieicent matrix 
* @param rows : rows,cols of coeff matrix
* @param maxIt : Maximum iterations*/

void LinearSolve( int* rowPtr, int* colPtr, double* val, 
                  double* Soln, double* RHS,                   
                  int rows, int nz, int maxit ){



    /*Variables to hold the info for BSR representation*/
    double*val_BSR = NULL;  //Values in BSR
    double* d_Y, *d_X;      //Soln and RHS resized for BSR dimensions
    int* row_BSR, *col_BSR; // Row offsets and col Indices in BSR
    int dimBlock = 2;       //Dim of BSR block - hardcoded
    int mb = (rows + dimBlock-1)/dimBlock;      //Resized no of rows in BSR
    int nb = (rows + dimBlock-1)/dimBlock;  //Resized no of cols in BSR
    int base; 
    int vecSize = rows + dimBlock-1; //Resized Soln dimension - BSR
    checkCudaErrors(hipMalloc((double **)&d_X, (vecSize)*sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&d_Y, (vecSize)*sizeof(double)));


    /*Part 1 - Convert input from CSR to BSR format*/
    int bufferSize;
    void *pBuffer;
    int nnzb = 0; 
    hipsparseHandle_t handle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&handle);
    
    hipblasHandle_t handleBlas = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&handleBlas);

    const hipsparseOperation_t trans_coeff  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseDirection_t dir_coeff = HIPSPARSE_DIRECTION_ROW;

    hipsparseMatDescr_t descr_coeff;
    hipsparseMatDescr_t descr_coeff_2;
    checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff));
    checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff, HIPSPARSE_INDEX_BASE_ZERO));
    checkCudaErrors(hipsparseSetMatType(descr_coeff, HIPSPARSE_MATRIX_TYPE_GENERAL));

    checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff_2));
    checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff_2, HIPSPARSE_INDEX_BASE_ZERO));
    checkCudaErrors(hipsparseSetMatType(descr_coeff_2, HIPSPARSE_MATRIX_TYPE_GENERAL));


    checkCudaErrors(hipMalloc((void**)&row_BSR, sizeof(int) *(mb+1)));


    /*Reserve buffer space and obtain number of nonzero blocks in BSR*/
    pBuffer =  getBSRDims(handle, rowPtr, colPtr, val, row_BSR, &nnzb,
                    rows, dimBlock); 


    /*Allocate cols and vals based on nonzero blocks*/
    checkCudaErrors(hipMalloc((void**)&col_BSR, sizeof(int)*(nnzb)));
    checkCudaErrors(hipMalloc((void**)&val_BSR, sizeof(double)*(dimBlock*dimBlock)*(nnzb)));

    /*Convert CSR to BSR*/
    checkCudaErrors(hipsparseDcsr2gebsr(handle, dir_coeff,rows, rows, descr_coeff, 
       val, rowPtr, colPtr, descr_coeff_2, val_BSR, row_BSR, col_BSR,        
        dimBlock, dimBlock, pBuffer));

    /*Transfer RHS and Soln to resized vectors*/
    checkCudaErrors(hipblasDcopy(handleBlas,rows, RHS, 1, d_X, 1));
    checkCudaErrors(hipblasDcopy(handleBlas, rows, Soln, 1, d_Y,1));

    /*Part 2 : Perform BCG solve*/
    BCGSolve(d_Y,  d_X,  row_BSR,  col_BSR, val_BSR, 
              handle, descr_coeff_2, handleBlas, 
               mb, nb, nnzb, vecSize, maxit); 

    /*Transfer result back to Soln*/
    checkCudaErrors(hipblasDcopy(handleBlas, rows, d_Y, 1, Soln,1));

    checkCudaErrors(hipFree(pBuffer));

}
